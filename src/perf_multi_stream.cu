#include <iostream>
#include <thrust/device_vector.h>

#include <cupti.h>

#include"multistream_scheduler.cuh"
#include "flashinfer_ops.cuh"

using flashinfer::PosEncodingMode;
using flashinfer::QKVLayout;

struct single_decode_input_data {
  size_t seq_len = 8192;
  size_t num_qo_heads = 32;
  size_t num_kv_heads = 32;
  size_t head_dim = 128;
  size_t pos_encoding_mode = 0;
  size_t kv_layout = 0;
  bool cooperative = true;

  thrust::device_vector<half>* Q = nullptr;
  thrust::device_vector<half>* K = nullptr;
  thrust::device_vector<half>* V = nullptr;
  thrust::device_vector<half>* O = nullptr;
  thrust::device_vector<half>* tmp = nullptr;
  // Allocate input data:
  single_decode_input_data() {
    Q = new thrust::device_vector<half>(num_qo_heads * head_dim);
    K = new thrust::device_vector<half>(seq_len * num_kv_heads * head_dim);
    V = new thrust::device_vector<half>(seq_len * num_kv_heads * head_dim);
    O = new thrust::device_vector<half>(num_qo_heads * head_dim);
    tmp = new thrust::device_vector<half>(16 * 1024 * 1024);
  }

  ~single_decode_input_data() {
    delete tmp;
    delete O;
    delete V;
    delete K;
    delete Q;
  }
};

struct single_prefill_input_data {
  size_t kv_len = 8192;
  size_t qo_len = kv_len;
  size_t num_kv_heads = 32;
  size_t num_qo_heads = 32;
  size_t head_dim = 128;
  size_t pos_encoding_mode = 0;
  size_t kv_layout = 0;
  bool causal = false;
  bool cooperative = true;
  bool allow_fp16_qk_reduction = false;

  // Allocate input data:
  thrust::device_vector<half>* Q = nullptr;
  thrust::device_vector<half>* K = nullptr;
  thrust::device_vector<half>* V = nullptr;
  thrust::device_vector<uint8_t>* mask = nullptr;
  thrust::device_vector<half>* O = nullptr;
  thrust::device_vector<half>* tmp = nullptr;

  single_prefill_input_data() {
    Q = new thrust::device_vector<half>(qo_len * num_qo_heads * head_dim);
    K = new thrust::device_vector<half>(kv_len * num_kv_heads * head_dim);
    V = new thrust::device_vector<half>(kv_len * num_kv_heads * head_dim);
    mask = new thrust::device_vector<uint8_t>(qo_len * kv_len / 8);
    O = new thrust::device_vector<half>(qo_len * num_qo_heads * head_dim);
    tmp = new thrust::device_vector<half>(16 * 1024 * 1024);
  }

  ~single_prefill_input_data() {
    delete tmp;
    delete O;
    delete mask;
    delete V;
    delete K;
    delete Q;
  }
};

void perf_flashinfer_single_decode(hipStream_t& stream, single_decode_input_data* input) {
  // Provide throughput information:
  hipError_t status = flashinfer::SingleDecodeWithKVCache(
      thrust::raw_pointer_cast(input->Q->data()), thrust::raw_pointer_cast(input->K->data()),
      thrust::raw_pointer_cast(input->V->data()), thrust::raw_pointer_cast(input->O->data()),
      input->cooperative ? thrust::raw_pointer_cast(input->tmp->data()) : nullptr, input->num_qo_heads, input->num_kv_heads,
      input->seq_len, input->head_dim, QKVLayout(input->kv_layout), PosEncodingMode(input->pos_encoding_mode),
      /*maybe_sm_scale=*/std::nullopt,
      /*rope_scale=*/1.f,
      /*rope_theta=*/1e4, stream);
  if (status != hipSuccess) {
    std::cout << "Execution error" << std::endl;
  }
}

void perf_flashinfer_single_prefill(hipStream_t& stream, single_prefill_input_data* input) {
  auto status = flashinfer::SinglePrefillWithKVCache<half, half>(
      thrust::raw_pointer_cast(input->Q->data()), thrust::raw_pointer_cast(input->K->data()),
      thrust::raw_pointer_cast(input->V->data()), thrust::raw_pointer_cast(input->O->data()),
      input->cooperative ? thrust::raw_pointer_cast(input->tmp->data()) : nullptr,
      nullptr, input->num_qo_heads, input->num_kv_heads, input->qo_len, input->kv_len, input->head_dim,
      input->causal, QKVLayout(input->kv_layout), PosEncodingMode(input->pos_encoding_mode),
      input->allow_fp16_qk_reduction, std::nullopt, 1.f, 1e4, stream);

  if (status != hipSuccess) {
    std::cout << "Execution error" << std::endl;
  }
}

int main() {
  const int numGPUs = 4;
  std::vector<single_decode_input_data*> decode_data;
  std::vector<single_prefill_input_data*> prefill_data;
  for(int i = 0;i < numGPUs;i ++){
    hipSetDevice(i);
    decode_data.push_back(new single_decode_input_data());
    prefill_data.push_back(new single_prefill_input_data());
  }

  const int iter = 100;

  {
    printf("========== one gpu one stream performance ==========\n");
    Scheduler scheduler({1});
    int gpu;
    hipStream_t stream;
    scheduler.scheduleKernel(&gpu, &stream, false, ScheduleMode::FREE_MEMORY_SCHEDULE_MODE);
    // std::cout << "scheduled on gpu " << gpu << " stream " << stream << std::endl;

    hipEvent_t start, stop;
    hipEventCreate(&start);hipEventCreate(&stop);
    hipEventRecord(start, stream);
    for (int i = 0; i < iter; ++ i) {
      perf_flashinfer_single_prefill(stream, prefill_data[gpu]);
      for(int j = 0;j < 30;j ++){
        perf_flashinfer_single_decode(stream, decode_data[gpu]);
      }
    }
    hipDeviceSynchronize();

    hipEventRecord(stop, stream);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("elapsed time %f ms\n", milliseconds);

  }

  {
    printf("========== one gpu multi stream performance ==========\n");
    Scheduler scheduler({1});
    int gpu;
    hipStream_t tensor_stream;
    hipStream_t hip_stream;
  
    scheduler.scheduleGPU(&gpu, &hip_stream, &tensor_stream, ScheduleMode::FREE_MEMORY_SCHEDULE_MODE);
    // std::cout << "scheduled on gpu " << gpu << " stream " << tensor_stream << " " << hip_stream << std::endl;

    hipEvent_t start, stop;
    hipEventCreate(&start);hipEventCreate(&stop);
    hipEventRecord(start, tensor_stream);


    for (int i = 0; i < iter; ++ i) {
      perf_flashinfer_single_prefill(tensor_stream, prefill_data[gpu]);
      for(int j = 0;j < 30;j ++){
        perf_flashinfer_single_decode(hip_stream, decode_data[gpu]);
      }
    }
    hipDeviceSynchronize();

    hipEventRecord(stop, tensor_stream);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("elapsed time %f ms\n", milliseconds);


  }

  {
    printf("========== multi gpu one stream performance ==========\n");
    Scheduler scheduler({2, 3});

    int gpu;
    hipStream_t cudaStream, tensorStream;
    hipSetDevice(0);
    hipEvent_t start, stop;
    hipEventCreate(&start);hipEventCreate(&stop);
    hipEventRecord(start);
    for (int i = 0; i < iter; ++ i) {
      scheduler.scheduleGPU(&gpu, &cudaStream, &tensorStream, ScheduleMode::ROUND_ROBIN_SCHEDULE_MODE);
      // std::cout << "scheduled on gpu " << gpu << " stream " << cudaStream << std::endl;

      perf_flashinfer_single_prefill(cudaStream, prefill_data[gpu]);
      perf_flashinfer_single_decode(cudaStream, decode_data[gpu]);
    }

    for(int i = 0;i < 4;i ++){
      hipSetDevice(i);
      hipDeviceSynchronize();
    }

    hipSetDevice(0);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("elapsed time %f ms\n", milliseconds);
  }

  {
    printf("========== multi gpu multi stream performance ==========\n");
    Scheduler scheduler({2, 3});

    int gpu;
    hipStream_t cudaStream, tensorStream;
    hipSetDevice(0);
    hipEvent_t start, stop;
    hipEventCreate(&start);hipEventCreate(&stop);
    hipEventRecord(start);
    for (int i = 0; i < iter; ++ i) {
      scheduler.scheduleGPU(&gpu, &cudaStream, &tensorStream, ScheduleMode::ROUND_ROBIN_SCHEDULE_MODE);
      // std::cout << "scheduled on gpu " << gpu << " stream " << cudaStream << " " << tensorStream << std::endl;

      perf_flashinfer_single_prefill(tensorStream, prefill_data[gpu]);
      perf_flashinfer_single_decode(cudaStream, decode_data[gpu]);
    }

    for(int i = 0;i < 4;i ++){
      hipSetDevice(i);
      hipDeviceSynchronize();
    }

    hipSetDevice(0);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("elapsed time %f ms\n", milliseconds);
  }
}